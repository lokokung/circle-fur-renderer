#include "hip/hip_runtime.h"
#include "gpu_raytracer.cuh"

__global__ void raytrace_base_kernel(Camera* camera, Light* light,
                                     Sphere* sphere, int xres, int yres,
                                     Vec3f *screen) {
    // Get current thread id
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    // Verify that we are within the image
    if (index < xres * yres - 1) {

        // Define maximum color
        Eigen::Vector3f max_color;
        max_color << 1.0, 1.0, 1.0;

        // Get camera information
        float near, aspect, fov, h, w;
        near = camera->near;
        aspect = camera->aspect;
        fov = camera->fov;
        h = 2 * near * tanf(fov * M_PI / 360.0);
        w = aspect * h;

        // Make basis vectors
        Eigen::Vector3f e1, e2, e3;
        e1 << 0.0, 0.0, -1.0;
        e2 << 1.0, 0.0, 0.0;
        e3 << 0.0, 1.0, 0.0;

        // Get camera position
        Eigen::Vector3f cp = camera->position;
        // Get position of the sphere
        Eigen::Vector3f sp = sphere->position;
        // Get color vectors of the sphere
        Eigen::Vector3f diffuse, specular;
        float p = sphere->p;
        diffuse = sphere->diffuse;
        specular = sphere->specular;

        // Convert id into x and y in the pixel grid
        int i = index % xres;
        int j = index / xres;

        // Define xi and yj
        float xi = (i - xres / 2.0) * (w / xres);
        float yj = (j - yres / 2.0) * (h / yres);
        // Compute direction of the camera ray
        Eigen::Vector3f direction =
            (near * e1) + (xi * e2) + (yj * e3);
        direction = direction / direction.norm();

        // Compute intersection between sphere and camera ray
        float t_minus, t_plus;
        int res = sphere->intersect_base(cp, direction,
                                         &t_minus, &t_plus);
            
        // Only continue if we actually intersected with something
        if (res) {

            // Initialize colors for the algorithm
            Eigen::Vector3f ldiffuse, lspecular;
            ldiffuse << 0.0, 0.0, 0.0;
            lspecular << 0.0, 0.0, 0.0;

        
            // Get this lights information
            Eigen::Vector3f lp = light->position;
            Eigen::Vector3f lc = light->color;
            float k = light->k;
                
            // Compute point of intersection and normal
            Eigen::Vector3f point = cp + t_minus * direction;
            Eigen::Vector3f normal = point - sp;
            normal = normal / normal.norm();

            // Get light distance
            Eigen::Vector3f ld = lp - point;
            float distance = ld.norm();
            ld = ld / ld.norm();

            // Apply attenuation
            Eigen::Vector3f lc_atten =
                            lc / (1.0 + k * distance * distance);
                
            // Compute diffuse term for lighting
            ldiffuse +=
                lc * fmaxf((float) 0.0, normal.dot(ld));

            // Compute specular term for lighting
            Eigen::Vector3f edld = ld - direction;
            edld = edld / edld.norm();
            lspecular +=
                lc * powf(fmaxf((float) 0.0,
                                normal.dot(edld)), p);
                                   
            // Compute total color
            Eigen::Vector3f color =
                diffuse.cwiseProduct(ldiffuse) +
                specular.cwiseProduct(lspecular);
            color = color.cwiseMin(max_color);
            
            // Set the png color
            screen[index].x += color(0, 0);
            screen[index].y += color(1, 0);
            screen[index].z += color(2, 0);
        }
    }
}
                                     
void GPURaytracer::cuda_raytrace_base(Scene* scene, int xres, int yres,
                                      Vec3f* screen, int n_threads) {
    // Compute kernel parameters
    int n_blocks = ((xres * yres) / n_threads) + 1;
    
    // Declare GPU side data
    Camera* camera;
    Light* light;
    Sphere* sphere;
    Vec3f* dev_screen;
    
    // Allocate GPU memory
    hipMalloc((void**) &camera, sizeof(Camera));
    hipMalloc((void**) &light, sizeof(Light));
    hipMalloc((void**) &sphere, sizeof(Sphere));
    hipMalloc((void**) &dev_screen, xres * yres * sizeof(Vec3f));

    // Initialize GPU memory
    hipMemcpy(camera, scene->camera, sizeof(Camera),
               hipMemcpyDefault);
    hipMemcpy(sphere, scene->sphere, sizeof(Sphere),
               hipMemcpyDefault);
    hipMemset(dev_screen, 0, xres * yres * sizeof(Vec3f));

    // Iterate over each light
    for (Light* l : scene->lights) {
        // Copy light over to GPU
        hipMemcpy(light, l, sizeof(Light),
                   hipMemcpyDefault);

        // Call kernel to add this lights effects
        raytrace_base_kernel<<<n_blocks, n_threads>>>(camera, light, sphere,
                                                      xres, yres, dev_screen);
    }

    // Copy screen data back to CPU
    hipMemcpy(screen, dev_screen, xres * yres * sizeof(Vec3f),
               hipMemcpyDefault);

    // Clean up GPU memory
    hipFree(camera);
    hipFree(light);
    hipFree(sphere);
    hipFree(dev_screen);
}


__global__ void raytrace_hair_init(GPUFurUtils** futils,
                                   hiprandState* states, int seed,
                                   NoiseModule** modules,
                                   int* module_seeds,
                                   float* bases,
                                   int xres, int yres) {

    // Get current thread id
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    // Set up CUDA random library states
    if (index < xres * yres - 1) {
        hiprand_init(seed, index, 0, &states[index]);
    }
        
    // Use first thread only to set up other state information
    if (index == 0) {
        // Initialize utilities
        *futils = new GPUFurUtils();
        
        // Set bases location and modules location
        (*futils)->set_noise_bases(bases);
        (*futils)->set_noise_modules(modules);
        
        // Initialize noise modules with seeds
        for (int i = 0; i < N_NOISE_TYPE; ++i) {
            modules[i] = new Perlin();
            ((Perlin*) modules[i])->SetSeed(module_seeds[i]);
        }
    }
}

__global__ void raytrace_hair_clean(GPUFurUtils** futils,
                                    NoiseModule** modules) {
    // Free memory used for the utilities
    for (int i = 0; i < N_NOISE_TYPE; ++i) {
        delete modules[i];
    }
    delete (*futils);
}

__global__ void raytrace_hair_kernel(Camera* camera, Light* light,
                                     Sphere* sphere, GPUFurUtils** futils,
                                     hiprandState* states,
                                     int xres, int yres,
                                     Vec3f *screen) {
    // Get current thread id
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    // Verify that we are within the image
    if (index < xres * yres - 1) {
        
        // Define maximum color
        Eigen::Vector3f max_color;
        max_color << 1.0, 1.0, 1.0;

        // Get camera information
        float near, aspect, fov, h, w;
        near = camera->near;
        aspect = camera->aspect;
        fov = camera->fov;
        h = 2 * near * tanf(fov * M_PI / 360.0);
        w = aspect * h;

        // Make basis vectors
        Eigen::Vector3f e1, e2, e3;
        e1 << 0.0, 0.0, -1.0;
        e2 << 1.0, 0.0, 0.0;
        e3 << 0.0, 1.0, 0.0;

        // Get camera position
        Eigen::Vector3f cp = camera->position;
        // Get position of the sphere
        Eigen::Vector3f sp = sphere->position;
        // Get color vectors of the sphere
        Eigen::Vector3f diffuse, specular;
        float p = sphere->p;
        diffuse = sphere->diffuse;
        specular = sphere->specular;

        // Convert id into x and y in the pixel grid
        int i = index % xres;
        int j = index / xres;

        // Define xi and yj
        float xi = (i - xres / 2.0) * (w / xres);
        float yj = (j - yres / 2.0) * (h / yres);
        // Compute direction of the camera ray
        Eigen::Vector3f direction =
            (near * e1) + (xi * e2) + (yj * e3);
        direction = direction / direction.norm();

        // Compute intersection between sphere and camera ray
        float t_minus_base = FLT_MAX;
        float t_plus_base = FLT_MAX;
        sphere->intersect_base(cp, direction,
                               &t_minus_base,
                               &t_plus_base);
        // Compute intersection with hair volume
        float t_minus_hair = FLT_MAX;
        float t_plus_hair = FLT_MAX;
        int res = sphere->intersect_hair(cp, direction,
                                         &t_minus_hair,
                                         &t_plus_hair);
            
        // Only continue if we actually intersected with some hair
        if (res > 1) {
            // Compute points of intersection with hair volume
            float t_min = t_minus_hair;
            float t_max = min(t_minus_base, t_plus_hair);
            
            // Initialize colors for the algorithm
            Eigen::Vector3f ldiffuse, lspecular;
            ldiffuse << 0.0, 0.0, 0.0;
            lspecular << 0.0, 0.0, 0.0;
            
            // Initialize density summation
            float sum_rho = 0.0;

            // Get this lights information
            Eigen::Vector3f lp = light->position;
            Eigen::Vector3f lc = light->color;
            float k = light->k;
                
            // Iterate steps to compute light model for hairs
            for (float t_low = t_min; t_low < t_max; t_low += STEP_SIZE) {
                // Compute upper range for segment
                float t_high = min(t_low + (float) STEP_SIZE, t_max);
                
                // Get random distance for Monte Carlo integration
                float t_rand = CUDA_RANDF(t_low, t_high, states + index);

                // Compute point on intersecting ray
                Eigen::Vector3f t_point = cp + t_rand * direction;
                // Compute normal
                Eigen::Vector3f t_b = t_point - sp;
                t_b /= t_b.norm();
                // Compute point on sphere
                Eigen::Vector3f t_base = t_b * sphere->radius;
                // Compute vector from base to light and distance
                Eigen::Vector3f t_l = lp - t_base;
                float distance = t_l.norm();
                t_l /= t_l.norm();
                // Compute vector from base to camera
                Eigen::Vector3f t_e = cp - t_base;
                t_e /= t_e.norm();

                // Apply light distance attenuation
                Eigen::Vector3f lc_atten =
                    lc / (1.0 + k * distance * distance);

                // Compute density at the point
                float rho = (*futils)->fur_density(sphere, t_base,
                                                   TILING, EXPAND);
                rho *= (t_high - t_low) / ((float) STEP_SIZE);

                // Compute second light ray to handle shadowing
                float shadow = (*futils)->fur_shadow(sphere, t_point,
                                                     lp, &states[index],
                                                     TILING, EXPAND);

                // Compute transparency and increment density
                float t_t = expf(-sphere->hair_atten * sum_rho);
                sum_rho += rho;
                    
                // Compute scaled variables with density calculations
                float t_factors = t_t * rho * shadow;

                // Compute diffuse term
                ldiffuse += lc_atten * t_factors * t_b.cross(t_l).norm();

                // Compute specular term
                lspecular +=
                    lc_atten * t_factors *
                    powf(t_b.dot(t_l) *
                         t_b.dot(t_e) +
                         t_b.cross(t_l).norm() *
                         t_b.cross(t_e).norm(), p);
            }

            // Compute total color
            Eigen::Vector3f color =
                diffuse.cwiseProduct(ldiffuse) +
                specular.cwiseProduct(lspecular);
            color = color.cwiseMin(max_color);

            // Set the png color
            screen[index].x += color(0, 0);
            screen[index].y += color(1, 0);
            screen[index].z += color(2, 0);
        }
    }
}

void GPURaytracer::cuda_raytrace_hair(Scene* scene, int xres, int yres,
                                      Vec3f* screen, int n_threads) {
    // Compute kernel parameters
    int n_blocks = ((xres * yres) / n_threads) + 1;

    // Declare GPU side data
    Camera* camera;
    Light* light;
    Sphere* sphere;
    Vec3f* dev_screen;
    GPUFurUtils** futils;
    hiprandState* states;
    NoiseModule** modules;
    int* module_seeds;
    float* bases;
    
    // Allocate GPU memory
    hipMalloc((void**) &camera, sizeof(Camera));
    hipMalloc((void**) &light, sizeof(Light));
    hipMalloc((void**) &sphere, sizeof(Sphere));
    hipMalloc((void**) &dev_screen, xres * yres * sizeof(Vec3f));
    hipMalloc((void**) &futils, sizeof(GPUFurUtils*));
    hipMalloc((void**) &states, xres * yres * sizeof(hiprandState));
    hipMalloc((void**) &modules, N_NOISE_TYPE * sizeof(NoiseModule*));
    hipMalloc((void**) &module_seeds, N_NOISE_TYPE * sizeof(int));
    hipMalloc((void**) &bases, N_NOISE_TYPE * sizeof(float));

    // Initialize module seeds, and bases (temporarily hard-coded)
    int host_module_seeds[2] = {1, 2};
    hipMemcpy(module_seeds, host_module_seeds, N_NOISE_TYPE * sizeof(int),
               hipMemcpyDefault);
    float host_bases[2] = {0.123, 0.672};
    hipMemcpy(bases, host_bases, N_NOISE_TYPE * sizeof(float),
               hipMemcpyDefault);

    // Initialize scene information
    hipMemcpy(camera, scene->camera, sizeof(Camera),
               hipMemcpyDefault);
    hipMemcpy(sphere, scene->sphere, sizeof(Sphere),
               hipMemcpyDefault);
    hipMemset(dev_screen, 0, xres * yres * sizeof(Vec3f));

    // Initialize utilities and states (current seed hard-coded)
    raytrace_hair_init<<<n_blocks, n_threads>>>(futils, states, 1234,
                                                modules, module_seeds,
                                                bases, xres, yres);

    // Iterate over each light
    for (Light* l : scene->lights) {
        // Copy light over to GPU
        hipMemcpy(light, l, sizeof(Light),
                   hipMemcpyDefault);

        // Call kernel to add this lights effects
        raytrace_hair_kernel<<<n_blocks, n_threads>>>(camera, light, sphere,
                                                      futils, states,
                                                      xres, yres, dev_screen);
    }

    // Copy screen data back to CPU
    hipMemcpy(screen, dev_screen, xres * yres * sizeof(Vec3f),
               hipMemcpyDefault);

    // Clean up GPU memory
    raytrace_hair_clean<<<1, 1>>>(futils, modules);
    hipFree(camera);
    hipFree(light);
    hipFree(sphere);
    hipFree(dev_screen);
    hipFree(futils);
    hipFree(states);
    hipFree(modules);
    hipFree(module_seeds);
    hipFree(bases);
}
