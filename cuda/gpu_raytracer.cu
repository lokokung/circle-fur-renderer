#include "hip/hip_runtime.h"
#include "gpu_raytracer.cuh"

__global__ void raytrace_base_kernel(Camera* camera, Light* light,
                                     Sphere* sphere, int xres, int yres,
                                     Vec3f *screen) {
    // Get current thread id
    uint index = blockIdx.x * blockDim.x + threadIdx.x;

    // Verify that we are within the image
    if (index < xres * yres - 1) {

        // Define maximum color
        Eigen::Vector3f max_color;
        max_color << 1.0, 1.0, 1.0;

        // Get camera information
        float near, aspect, fov, h, w;
        near = camera->near;
        aspect = camera->aspect;
        fov = camera->fov;
        h = 2 * near * tan(fov * M_PI / 360.0);
        w = aspect * h;

        // Make basis vectors
        Eigen::Vector3f e1, e2, e3;
        e1 << 0.0, 0.0, -1.0;
        e2 << 1.0, 0.0, 0.0;
        e3 << 0.0, 1.0, 0.0;

        // Get camera position
        Eigen::Vector3f cp = camera->position;
        // Get position of the sphere
        Eigen::Vector3f sp = sphere->position;
        // Get color vectors of the sphere
        Eigen::Vector3f diffuse, specular;
        float p = sphere->p;
        diffuse = sphere->diffuse;
        specular = sphere->specular;

        // Convert id into x and y in the pixel grid
        int i = index % xres;
        int j = index / xres;

        // Define xi and yj
        float xi = (i - xres / 2.0) * (w / xres);
        float yj = (j - yres / 2.0) * (h / yres);
        // Compute direction of the camera ray
        Eigen::Vector3f direction =
            (near * e1) + (xi * e2) + (yj * e3);
        direction = direction / direction.norm();

        // Compute intersection between sphere and camera ray
        float t_minus, t_plus;
        int res = sphere->intersect_base(cp, direction,
                                         &t_minus, &t_plus);
            
        // Only continue if we actually intersected with something
        if (res) {

            // Initialize colors for the algorithm
            Eigen::Vector3f ldiffuse, lspecular;
            ldiffuse << 0.0, 0.0, 0.0;
            lspecular << 0.0, 0.0, 0.0;

        
            // Get this lights information
            Eigen::Vector3f lp = light->position;
            Eigen::Vector3f lc = light->color;
            float k = light->k;
                
            // Compute point of intersection and normal
            Eigen::Vector3f point = cp + t_minus * direction;
            Eigen::Vector3f normal = point - sp;
            normal = normal / normal.norm();

            // Get light distance
            Eigen::Vector3f ld = lp - point;
            float distance = ld.norm();
            ld = ld / ld.norm();

            // Apply attenuation
            Eigen::Vector3f lc_atten =
                            lc / (1.0 + k * distance * distance);
                
            // Compute diffuse term for lighting
            ldiffuse +=
                lc * fmaxf((float) 0.0, normal.dot(ld));

            // Compute specular term for lighting
            Eigen::Vector3f edld = ld - direction;
            edld = edld / edld.norm();
            lspecular +=
                lc * powf(fmaxf((float) 0.0,
                                normal.dot(edld)), p);
                                   
            // Compute total color
            Eigen::Vector3f color =
                diffuse.cwiseProduct(ldiffuse) +
                specular.cwiseProduct(lspecular);
            color = color.cwiseMin(max_color);
            
            // Set the png color
            screen[index].x += color(0, 0);
            screen[index].y += color(1, 0);
            screen[index].z += color(2, 0);
        }
    }
}
                                     
void GPURaytracer::cuda_raytrace_base(Scene* scene, int xres, int yres,
                                      Vec3f* screen, int n_threads) {
    // Compute kernel parameters
    int n_blocks = ((xres * yres) / n_threads) + 1;
    
    // Declare GPU side data
    Camera* camera;
    Light* light;
    Sphere* sphere;
    Vec3f* dev_screen;
    
    // Allocate GPU memory
    hipMalloc((void**) &camera, sizeof(Camera));
    hipMalloc((void**) &light, sizeof(Light));
    hipMalloc((void**) &sphere, sizeof(Sphere));
    hipMalloc((void**) &dev_screen, xres * yres * sizeof(Vec3f));

    // Initialize GPU memory
    hipMemcpy(camera, scene->camera, sizeof(Camera),
               hipMemcpyDefault);
    hipMemcpy(sphere, scene->sphere, sizeof(Sphere),
               hipMemcpyDefault);
    hipMemset(dev_screen, 0, xres * yres * sizeof(Vec3f));

    // Iterate over each light
    for (Light* l : scene->lights) {
        // Copy light over to GPU
        hipMemcpy(light, l, sizeof(Light),
                   hipMemcpyDefault);

        // Call kernel to add this lights effects
        raytrace_base_kernel<<<n_blocks, n_threads>>>(camera, light, sphere,
                                                      xres, yres, dev_screen);
    }

    // Copy screen data back to CPU
    hipMemcpy(screen, dev_screen, xres * yres * sizeof(Vec3f),
               hipMemcpyDefault);

    // Clean up GPU memory
    hipFree(camera);
    hipFree(light);
    hipFree(sphere);
    hipFree(dev_screen);
}
