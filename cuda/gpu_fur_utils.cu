#include "hip/hip_runtime.h"
#include "gpu_fur_utils.cuh"

GPUFurUtils::GPUFurUtils() {
    state = NULL;
}

void GPUFurUtils::set_noise_state(hiprandState* state) {
    this->state = state;
}

void GPUFurUtils::set_noise_modules(NoiseModule** modules) {
    this->modules = modules;
}

void GPUFurUtils::set_noise_bases(float* bases) {
    this->bases = bases;
}

__device__ float GPUFurUtils::fur_density(Sphere* sphere, Eigen::Vector3f point,
                                           int tiling, float expand) {
    // Compute spherical coordinates
    float theta, phi;
    sphere->spherical_coord(point, &theta, &phi);

    // Compute u and v for mapping
    float u = (tiling * M_PI) * theta / (2.0 * M_PI);
    float v = (tiling * M_PI) * (cosf(phi) + 1) / 2.0;

    // Compute noise per hair
    float ui = ((int) (u / M_PI)) + bases[U_NOISE];
    float vi = ((int) (v / M_PI)) + bases[V_NOISE];
    float du = (M_PI / 4.0) *
        ((Perlin*) modules[U_NOISE])->GetValue(ui, vi, bases[U_NOISE]);
    float dv = (M_PI / 4.0) *
        ((Perlin*) modules[V_NOISE])->GetValue(ui, vi, bases[V_NOISE]);
    
    // Compute density at this point
    float rho =
        sinf(u + du) / 2.0 + sinf(v + dv) / 2.0;
    rho = powf(rho, expand);

    return rho;
}

__device__ float GPUFurUtils::fur_shadow(Sphere* sphere, Eigen::Vector3f point,
                                         Eigen::Vector3f lp, hiprandState* state,
                                         int tiling, float expand) {
    // Compute t for the actual point and direction
    Eigen::Vector3f direction = point - lp;
    float t = direction.norm();
    direction /= t;

    // Intersect ray with sphere
    float t_minus_base = FLT_MAX;
    float t_plus_base  = FLT_MAX;
    sphere->intersect_base(lp, direction,
                           &t_minus_base,
                           &t_plus_base);
    // Intersect ray with hairs
    float t_minus_hair = FLT_MAX;
    float t_plus_hair = FLT_MAX;
    int res = sphere->intersect_hair(lp, direction,
                                     &t_minus_hair,
                                     &t_plus_hair);

    // Check that we intersected (should be trivially true) and then
    // iterate across the ray to get sum of densities
    float sum_rho = 0.0;
    if (res > 1 && t_minus_hair < t && t_minus_base > t) {
        // Iterate across the ray
        for (float t_low = t_minus_hair; t_low < t; t_low += STEP_SIZE) {
            // Compute upper range for segment
            float t_high = min(t_low + (float) STEP_SIZE, t);

            // Get random distance for Monte Carlo integration
            float t_rand = CUDA_RANDF(t_low, t_high, state);

            // Compute point on intersecting ray
            Eigen::Vector3f t_point = lp + t_rand * direction;
            // Compute normal
            Eigen::Vector3f t_b = t_point - sphere->position;
            t_b /= t_b.norm();
            // Compute point on sphere
            Eigen::Vector3f t_base = t_b * sphere->radius;

            // Compute density at the point
            float rho = fur_density(sphere, t_base,
                                    TILING, EXPAND);
            rho *= (t_high - t_low) / ((float) STEP_SIZE);

            // Add to total density
            sum_rho += rho;
        }
        
        // Compute transparency factor
        float transparency = expf(-sphere->hair_atten * sum_rho);
    
        return transparency;
    }

    // Otherwise, the light is completely blocked so return a factor of 0
    return 0.0;
}
